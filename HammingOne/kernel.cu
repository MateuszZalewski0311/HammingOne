// includes, system
#include <stdio.h>
#include <random>
#include <bitset>
#include <unordered_set>
#include <vector>
#include <chrono>
#include <algorithm>

// includes, cuda
#include "hip/hip_runtime.h"
#include ""

// includes, thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

////////////////////////////////////////////////////////////////////////////////
#define WORD_SIZE 23
#define DATA_SIZE 10000

////////////////////////////////////////////////////////////////////////////////
// function declarations
template<size_t N>
unsigned int hamming_distance(const typename std::bitset<N>& A, const typename std::bitset<N>& B);
template<size_t N>
typename std::bitset<N> random_bitset(double p);
template<size_t N, size_t M>
void generate_data(typename std::unordered_set<std::bitset<N>>& _data_uset, \
    const bool timeCount = true, const bool consoleOutput = true, const float p = 0.5f);
template<size_t N>
void find_ham1(const typename std::unordered_set<std::bitset<N>>& _data_uset, \
    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2, \
    const bool timeCount = true, const bool pairsOutput = true, const bool consoleOutput = true);
//template<size_t N>
//void find_ham1_temp(const typename std::unordered_set<std::bitset<N>>& _data_uset, \
//    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2, \
//    const bool timeCount = true, const bool pairsOutput = true, const bool consoleOutput = true);

////////////////////////////////////////////////////////////////////////////////
// word generating function
template<size_t N> // p = 0.5 gives equal chance for 0's and 1's to occur
typename std::bitset<N> random_bitset(double p) 
{
    typename std::bitset<N> bits;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::bernoulli_distribution dist(p);

    for (size_t i = 0; i < N; ++i) {
        bits[i] = dist(gen);
    }

    return bits;
}

////////////////////////////////////////////////////////////////////////////////
// data generating function
template<size_t N, size_t M>
void generate_data(typename std::unordered_set<std::bitset<N>>& _data_uset, \
    const bool timeCount, const bool consoleOutput, const float p)
{
    std::chrono::steady_clock::time_point start, finish;
    std::chrono::duration<double> elapsed;

    if (consoleOutput) std::cout << "Beginning Data Generation...\n";

    // Record start time
    if (consoleOutput && timeCount) start = std::chrono::high_resolution_clock::now();

    for (size_t i = 0; i < M; ++i)
    {
        while (false == (_data_uset.emplace(random_bitset<N>(p)).second));
    }

    // Record end time
    if (consoleOutput && timeCount) finish = std::chrono::high_resolution_clock::now();

    if (consoleOutput)
    {
        if (timeCount) elapsed = finish - start;
        std::cout << "Data Generation Finished!\n";
        if (timeCount) std::cout << "Elapsed time: " << elapsed.count() << " s\n";
        //std::cout << "Data has " << data.size() << " unique elements\n";
        std::cout << std::endl;
    }
}

////////////////////////////////////////////////////////////////////////////////
// data loading function
//
////////////////////////////////////////////////////////////////////////////////
// data saving function

////////////////////////////////////////////////////////////////////////////////
// finding pairs with hamming distance 1 function
template<size_t N>
void find_ham1(const typename std::unordered_set<std::bitset<N>>& _data_uset, \
    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2, \
    const bool timeCount, const bool pairsOutput, const bool consoleOutput)
{
    std::chrono::steady_clock::time_point start, finish;
    std::chrono::duration<double> elapsed;

    if (consoleOutput) std::cout << "Looking for pairs with hamming distance 1 ...\n";

    // Record start time
    if (consoleOutput && timeCount) start = std::chrono::high_resolution_clock::now();

    unsigned int ham1 = 0;
    for (auto it1 = std::begin(_data_uset); it1 != std::end(_data_uset); ++it1)
    {
        for (auto it2 = std::next(it1); it2 != std::end(_data_uset); ++it2)
        {
            if (1 == hamming_distance<N>(*it1, *it2))
            {
                _ham1_pairs_1.emplace_back(std::bitset<N>(*it1));
                _ham1_pairs_2.emplace_back(std::bitset<N>(*it2));
                //_ham1_pairs_1.push_back(*it1);
                //_ham1_pairs_2.push_back(*it2);
                ++ham1;
            }
        }
    }

    // Record end time
    if (consoleOutput && timeCount) finish = std::chrono::high_resolution_clock::now();

    if (consoleOutput)
    {
        if (timeCount) elapsed = finish - start;
        std::cout << "Finished!\n";
        if (timeCount) std::cout << "Elapsed time: " << elapsed.count() << " s\n";
        std::cout << ham1 << " pairs found\n\n";
    }

    if (ham1 && pairsOutput && consoleOutput)
    {
        std::cout << "Pairs found:\n";

        for (auto it1 = std::begin(_ham1_pairs_1), it2 = std::begin(_ham1_pairs_2); it1 != std::end(_ham1_pairs_1); ++it1, ++it2)
        {
            std::cout << *it1 << " " << *it2 << std::endl;
        }

        std::cout << std::endl;
    }
}

//template<size_t N>
//void find_ham1_temp(const typename std::unordered_set<std::bitset<N>>& _data_uset, \
//    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2, \
//    const bool timeCount, const bool pairsOutput, const bool consoleOutput)
//{
//    std::chrono::steady_clock::time_point start, finish;
//    std::chrono::duration<double> elapsed;
//
//    if (consoleOutput) std::cout << "Looking for pairs with hamming distance 1 ...\n";
//
//    // Record start time
//    if (consoleOutput && timeCount) start = std::chrono::high_resolution_clock::now();
//
//    unsigned int ham1 = 0;
//    for (const auto& A : _data_uset)
//    {
//        for (const auto& B : _data_uset)
//        {
//            if (1 == hamming_distance<N>(A, B))
//            {
//                auto it1 = std::find(std::begin(_ham1_pairs_2), std::end(_ham1_pairs_2), A);
//                auto it2 = std::find(std::begin(_ham1_pairs_1), std::end(_ham1_pairs_1), B);
//                if (it1 != std::end(_ham1_pairs_2) && it2 != std::end(_ham1_pairs_1) && it1 - std::begin(_ham1_pairs_2) == it2 - std::begin(_ham1_pairs_1)) {
//                    continue;
//                }
//                _ham1_pairs_1.emplace_back(std::bitset<N>(A));
//                _ham1_pairs_2.emplace_back(std::bitset<N>(B));
//                ++ham1;
//            }
//        }
//    }
//
//    // Record end time
//    if (consoleOutput && timeCount) finish = std::chrono::high_resolution_clock::now();
//
//    if (consoleOutput)
//    {
//        if (timeCount) elapsed = finish - start;
//        std::cout << "Finished!\n";
//        if (timeCount) std::cout << "Elapsed time: " << elapsed.count() << " s\n";
//        std::cout << ham1 << " pairs found\n\n";
//    }
//
//    if (ham1 && pairsOutput && consoleOutput)
//    {
//        std::cout << "Pairs found:\n";
//
//        for (auto it1 = std::begin(_ham1_pairs_1), it2 = std::begin(_ham1_pairs_2); it1 != std::end(_ham1_pairs_1); ++it1, ++it2)
//        {
//            std::cout << *it1 << " " << *it2 << std::endl;
//        }
//
//        std::cout << std::endl;
//    }
//}

////////////////////////////////////////////////////////////////////////////////
// hamming distance function
template<size_t N>
unsigned int hamming_distance(const typename std::bitset<N>& A, const typename std::bitset<N>& B)
{
    return (A ^ B).count();
}

////////////////////////////////////////////////////////////////////////////////
int main()
{
    //hipError_t cudaStatus;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/

    //thrust::host_vector<std::bitset<1024>> h_data(100000);
    std::unordered_set<std::bitset<WORD_SIZE>> data_uset;
    std::vector<std::bitset<WORD_SIZE>> ham1_pairs_1;
    std::vector<std::bitset<WORD_SIZE>> ham1_pairs_2;

    data_uset.reserve(DATA_SIZE);
    generate_data<WORD_SIZE, DATA_SIZE>(data_uset);

    find_ham1<WORD_SIZE>(data_uset, ham1_pairs_1, ham1_pairs_2, true, true);

    system("pause");

    return 0;
}

