#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <random>
#include <bitset>
#include <unordered_set>
#include <vector>
#include <chrono>
#include <algorithm>
#include <limits>

// includes, cuda
#include "hip/hip_runtime.h"
#include ""

// includes, thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

////////////////////////////////////////////////////////////////////////////////
#define WORD_SIZE 23
#define DATA_SIZE 10000

////////////////////////////////////////////////////////////////////////////////
// function declarations
template<size_t N>
unsigned int hamming_distance(const typename std::bitset<N>& A, const typename std::bitset<N>& B);
template<size_t N>
typename std::bitset<N> random_bitset(double p);
template<size_t N, size_t M>
void generate_data(typename std::unordered_set<std::bitset<N>>& _data_uset, \
    const bool timeCount = true, const bool consoleOutput = true, const float p = 0.5f);
template<size_t N>
void find_ham1(const typename std::unordered_set<std::bitset<N>>& _data_uset, \
    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2, \
    const bool timeCount = true, const bool pairsOutput = true, const bool consoleOutput = true);
//template<size_t N>
//void find_ham1_temp(const typename std::unordered_set<std::bitset<N>>& _data_uset, \
//    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2, \
//    const bool timeCount = true, const bool pairsOutput = true, const bool consoleOutput = true);

////////////////////////////////////////////////////////////////////////////////
// word generating function
template<size_t N> // p = 0.5 gives equal chance for 0's and 1's to occur
typename std::bitset<N> random_bitset(double p) 
{
    typename std::bitset<N> bits;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::bernoulli_distribution dist(p);

    for (size_t i = 0; i < N; ++i) {
        bits[i] = dist(gen);
    }

    return bits;
}

////////////////////////////////////////////////////////////////////////////////
// data generating function
template<size_t N, size_t M>
void generate_data(typename std::unordered_set<std::bitset<N>>& _data_uset, \
    const bool timeCount, const bool consoleOutput, const float p)
{
    std::chrono::steady_clock::time_point start, finish;
    std::chrono::duration<double> elapsed;

    if (consoleOutput) std::cout << "Beginning Data Generation...\n";

    // Record start time
    if (consoleOutput && timeCount) start = std::chrono::high_resolution_clock::now();

    for (size_t i = 0; i < M; ++i)
    {
        while (false == (_data_uset.emplace(random_bitset<N>(p)).second));
    }

    // Record end time
    if (consoleOutput && timeCount) finish = std::chrono::high_resolution_clock::now();

    if (consoleOutput)
    {
        if (timeCount) elapsed = finish - start;
        std::cout << "Data Generation Finished!\n";
        if (timeCount) std::cout << "Elapsed time: " << elapsed.count() << " s\n";
        //std::cout << "Data has " << data.size() << " unique elements\n";
        std::cout << std::endl;
    }
}

////////////////////////////////////////////////////////////////////////////////
// data loading function
//
////////////////////////////////////////////////////////////////////////////////
// data saving function

////////////////////////////////////////////////////////////////////////////////
// finding pairs with hamming distance 1 function
template<size_t N>
void find_ham1(const typename std::unordered_set<std::bitset<N>>& _data_uset, \
    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2, \
    const bool timeCount, const bool pairsOutput, const bool consoleOutput)
{
    std::chrono::steady_clock::time_point start, finish;
    std::chrono::duration<double> elapsed;

    if (consoleOutput) std::cout << "Looking for pairs with hamming distance 1 ...\n";

    // Record start time
    if (consoleOutput && timeCount) start = std::chrono::high_resolution_clock::now();

    unsigned int ham1 = 0;
    for (auto it1 = std::begin(_data_uset); it1 != std::end(_data_uset); ++it1)
    {
        for (auto it2 = std::next(it1); it2 != std::end(_data_uset); ++it2)
        {
            if (1 == hamming_distance<N>(*it1, *it2))
            {
                _ham1_pairs_1.emplace_back(std::bitset<N>(*it1));
                _ham1_pairs_2.emplace_back(std::bitset<N>(*it2));
                //_ham1_pairs_1.push_back(*it1);
                //_ham1_pairs_2.push_back(*it2);
                ++ham1;
            }
        }
    }

    // Record end time
    if (consoleOutput && timeCount) finish = std::chrono::high_resolution_clock::now();

    if (consoleOutput)
    {
        if (timeCount) elapsed = finish - start;
        std::cout << "Finished!\n";
        if (timeCount) std::cout << "Elapsed time: " << elapsed.count() << " s\n";
        std::cout << ham1 << " pairs found\n\n";
    }

    if (ham1 && pairsOutput && consoleOutput)
    {
        std::cout << "Pairs found:\n";

        for (auto it1 = std::begin(_ham1_pairs_1), it2 = std::begin(_ham1_pairs_2); it1 != std::end(_ham1_pairs_1); ++it1, ++it2)
        {
            std::cout << *it1 << " " << *it2 << std::endl;
        }

        std::cout << std::endl;
    }
}

//template<size_t N>
//void find_ham1_temp(const typename std::unordered_set<std::bitset<N>>& _data_uset, \
//    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2, \
//    const bool timeCount, const bool pairsOutput, const bool consoleOutput)
//{
//    std::chrono::steady_clock::time_point start, finish;
//    std::chrono::duration<double> elapsed;
//
//    if (consoleOutput) std::cout << "Looking for pairs with hamming distance 1 ...\n";
//
//    // Record start time
//    if (consoleOutput && timeCount) start = std::chrono::high_resolution_clock::now();
//
//    unsigned int ham1 = 0;
//    for (const auto& A : _data_uset)
//    {
//        for (const auto& B : _data_uset)
//        {
//            if (1 == hamming_distance<N>(A, B))
//            {
//                auto it1 = std::find(std::begin(_ham1_pairs_2), std::end(_ham1_pairs_2), A);
//                auto it2 = std::find(std::begin(_ham1_pairs_1), std::end(_ham1_pairs_1), B);
//                if (it1 != std::end(_ham1_pairs_2) && it2 != std::end(_ham1_pairs_1) && it1 - std::begin(_ham1_pairs_2) == it2 - std::begin(_ham1_pairs_1)) {
//                    continue;
//                }
//                _ham1_pairs_1.emplace_back(std::bitset<N>(A));
//                _ham1_pairs_2.emplace_back(std::bitset<N>(B));
//                ++ham1;
//            }
//        }
//    }
//
//    // Record end time
//    if (consoleOutput && timeCount) finish = std::chrono::high_resolution_clock::now();
//
//    if (consoleOutput)
//    {
//        if (timeCount) elapsed = finish - start;
//        std::cout << "Finished!\n";
//        if (timeCount) std::cout << "Elapsed time: " << elapsed.count() << " s\n";
//        std::cout << ham1 << " pairs found\n\n";
//    }
//
//    if (ham1 && pairsOutput && consoleOutput)
//    {
//        std::cout << "Pairs found:\n";
//
//        for (auto it1 = std::begin(_ham1_pairs_1), it2 = std::begin(_ham1_pairs_2); it1 != std::end(_ham1_pairs_1); ++it1, ++it2)
//        {
//            std::cout << *it1 << " " << *it2 << std::endl;
//        }
//
//        std::cout << std::endl;
//    }
//}

////////////////////////////////////////////////////////////////////////////////
// hamming distance function
template<size_t N>
unsigned int hamming_distance(const typename std::bitset<N>& A, const typename std::bitset<N>& B)
{
    return (A ^ B).count();
}

////////////////////////////////////////////////////////////////////////////////
int main()
{
    //hipError_t cudaStatus;

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }*/

    unsigned short menu_choice = 0;
    //thrust::host_vector<std::bitset<1024>> h_data(100000);
    std::unordered_set<std::bitset<WORD_SIZE>> data_uset;
    std::vector<std::bitset<WORD_SIZE>> ham1_pairs_1;
    std::vector<std::bitset<WORD_SIZE>> ham1_pairs_2;

    while (menu_choice != 4) {
        std::cout << "1. Generate Data" << std::endl;
        std::cout << "2. Save/Load Data" << std::endl;
        if (!data_uset.empty())
            std::cout << "3. Find Pairs" << std::endl;
        else
            std::cout << "3. Find Pairs - !!! Generate/Load Data before attempting to find pairs !!!" << std::endl;
        std::cout << "4. Exit" << std::endl;
        std::cout << "5. Clear Console" << std::endl;
        std::cout << "Choice: ";
        std::cin >> menu_choice;
        switch (menu_choice)
        {
        case 1:
            if (!data_uset.empty())
                data_uset.clear();
            data_uset.reserve(DATA_SIZE);
            std::cout << std::endl;
            generate_data<WORD_SIZE, DATA_SIZE>(data_uset);
            break;
        case 2:
            std::cout << std::endl << "Not implemented yet :(" << std::endl << std::endl;
            break;
        case 3:
            if (!data_uset.empty()) {
                std::cout << std::endl;
                char c;
                do {
                    std::cout << "Output pairs to console? (y/n):";
                    std::cin.clear();
                    std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
                    c = std::getc(stdin);
                    if (c == 'y' || c == 'Y') {
                        find_ham1<WORD_SIZE>(data_uset, ham1_pairs_1, ham1_pairs_2, true, true, true);
                        break;
                    }
                    else if (c == 'n' || c == 'N') {
                        find_ham1<WORD_SIZE>(data_uset, ham1_pairs_1, ham1_pairs_2, true, false, true);
                        break;
                    }
                    std::cout << "Please provide a valid choice" << std::endl;
                } while (true);
            }
            else
                std::cout << std::endl << "!!! Generate/Load Data before attempting to find pairs !!!" << std::endl << std::endl;
            break;
        case 4:
            break;
        case 5:
            system("CLS");
            break;
        default:
            std::cout << std::endl << "Please provide a valid choice" << std::endl << std::endl;;
            break;
        }
    }

    return 0;
}

