#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <random>
#include <bitset>
#include <unordered_set>
#include <vector>
#include <chrono>
#include <algorithm>
#include <limits>
#include <fstream>

// includes, cuda
#include "hip/hip_runtime.h"
#include ""

// includes, thrust
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

////////////////////////////////////////////////////////////////////////////////
#define WORD_SIZE 24
#define DATA_SIZE 10000
#define UINT_BITSIZE (unsigned int)(8*sizeof(unsigned int))
#define SUBWORDS_PER_WORD(N) (unsigned int)(std::ceil((float)N / (sizeof(unsigned int) * 8.0f)))

////////////////////////////////////////////////////////////////////////////////
// function declarations
template<size_t N>
unsigned int hamming_distance(const typename std::bitset<N>& A, const typename std::bitset<N>& B);
template<size_t N>
typename std::bitset<N> random_bitset(double p);
template<size_t N, size_t M>
void generate_data(typename std::vector<std::bitset<N>>& _data_vec, \
    const bool timeCount = true, const bool consoleOutput = true, const float p = 0.5f);
template<size_t N, size_t M>
void load_data(const char* words_filepath, const char* pairs_filepath, typename std::vector<std::bitset<N>>& _data_vec, \
    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2);
template<size_t N, size_t M>
void save_data(const char* words_filepath, const char* pairs_filepath, const typename std::vector<std::bitset<N>>& _data_vec, \
    const typename std::vector<std::bitset<N>>& _ham1_pairs_1, const typename std::vector<std::bitset<N>>& _ham1_pairs_2);
template<size_t N>
void find_ham1(const typename std::vector<std::bitset<N>>& _data_vec, \
    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2, \
    const bool timeCount = true, const bool pairsOutput = true);
template<size_t N, size_t M>
thrust::device_vector<unsigned int> move_data_to_GPU(const typename std::vector<std::bitset<N>>& data_vec);
__global__ void find_ham1_GPU_ker(const unsigned int* subwords, unsigned int* pair_flags, const unsigned int subwords_per_pair_flags);
//__global__ void count_ones(unsigned int* d_data, size_t pair_flags_size)
template<size_t N>
void find_ham1_GPU(thrust::device_vector<unsigned int>& d_subwords, \
    thrust::device_vector<unsigned int>& d_pair_flags, \
    thrust::host_vector<unsigned int>& h_pair_flags, size_t pair_flags_size, \
    const bool timeCount, const bool pairsOutput, const typename std::vector<std::bitset<N>>& _data_vec);
template<size_t N>
void print_pairs_from_flags(thrust::host_vector<unsigned int>& h_pair_flags, size_t pair_flags_size, \
    const typename std::vector<std::bitset<N>>& data_vec);

////////////////////////////////////////////////////////////////////////////////
// word generating function
template<size_t N> // p = 0.5 gives equal chance for 0's and 1's to occur
typename std::bitset<N> random_bitset(double p) 
{
    typename std::bitset<N> bits;
    std::random_device rd;
    std::mt19937 gen(rd());
    std::bernoulli_distribution dist(p);

    for (size_t i = 0; i < N; ++i) {
        bits[i] = dist(gen);
    }

    return bits;
}

////////////////////////////////////////////////////////////////////////////////
// data generating function
template<size_t N, size_t M>
void generate_data(typename std::vector<std::bitset<N>>& _data_vec, \
    const bool timeCount, const bool consoleOutput, const float p)
{
    std::unordered_set<std::bitset<N>> data_uset;
    data_uset.reserve(M);

    std::chrono::steady_clock::time_point start, finish;
    std::chrono::duration<double> elapsed;

    if (consoleOutput) std::cout << "Beginning Data Generation...\n";

    // Record start time
    if (consoleOutput && timeCount) start = std::chrono::high_resolution_clock::now();

    for (size_t i = 0; i < M; ++i)
    {
        while (false == (data_uset.emplace(random_bitset<N>(p)).second));
    }

    // Record end time
    if (consoleOutput && timeCount) finish = std::chrono::high_resolution_clock::now();

    // Copy to vector
    for (const auto& it : data_uset) 
    {
        _data_vec.emplace_back(it);
    }

    if (consoleOutput)
    {
        if (timeCount) elapsed = finish - start;
        std::cout << "Data Generation Finished!\n";
        if (timeCount) std::cout << "Elapsed time: " << elapsed.count() << " s\n";
        std::cout << std::endl;
    }
}

////////////////////////////////////////////////////////////////////////////////
// data loading function
template<size_t N, size_t M>
void load_data(const char* words_filepath, const char* pairs_filepath, typename std::vector<std::bitset<N>>& _data_vec, \
    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2)
{
    size_t pairs_count = 0;
    std::string line, number;
    std::string separator = ";";
    size_t sep_pos = 0;
    std::ifstream words_file;
    words_file.open(words_filepath);
    if (!words_file.good()) {
        std::cout << "Error opening words_file\n\n";
        return;
    }
    _data_vec.clear();
    std::getline(words_file, line);
    std::getline(words_file, line);
    sep_pos = line.find(separator);
    if (sep_pos == std::string::npos) {
        std::cout << "Error - wrong formatting\n\n";
        return;
    }
    if (std::stoi(line.substr(0, sep_pos)) != N) {
        std::cout << "Error - WORD_SIZE different\n\n";
        return;
    }
    if (std::stoi(line.substr(sep_pos + 1)) != M) {
        std::cout << "Error - DATA_SIZE different\n\n";
        return;
    }
    for (size_t i = 0; i < M; ++i) {
        std::getline(words_file, line);
        _data_vec.emplace_back(std::bitset<N>(line));
    }

    std::ifstream pairs_file;
    pairs_file.open(pairs_filepath);
    if (!words_file.good()) {
        std::cout << "Error opening pairs_file\n\n";
        return;
    }
    _ham1_pairs_1.clear();
    _ham1_pairs_2.clear();
    std::getline(pairs_file, line);
    std::getline(pairs_file, line);
    sep_pos = line.find(separator);
    if (sep_pos == std::string::npos) {
        std::cout << "Error - wrong formatting\n\n";
        return;
    }
    if (std::stoi(line.substr(0, sep_pos)) != N) {
        std::cout << "Error - WORD_SIZE different\n\n";
        return;
    }
    pairs_count = std::stoi(line.substr(sep_pos + 1));
    for (size_t i = 0; i < pairs_count; ++i) {
        std::getline(pairs_file, line);
        sep_pos = line.find(separator);
        _ham1_pairs_1.emplace_back(std::bitset<N>(line.substr(0, sep_pos)));
        _ham1_pairs_2.emplace_back(std::bitset<N>(line.substr(sep_pos + 1)));
    }
    pairs_file.close();

    std::cout << "Loading Data successful!" << std::endl << std::endl;
}
////////////////////////////////////////////////////////////////////////////////
// data saving function
template<size_t N, size_t M>
void save_data(const char* words_filepath, const char* pairs_filepath, const typename std::vector<std::bitset<N>>& _data_vec, \
    const typename std::vector<std::bitset<N>>& _ham1_pairs_1, const typename std::vector<std::bitset<N>>& _ham1_pairs_2)
{
    if (_data_vec.empty()) {
        std::cout << "Words vector is empty";
    }
    std::ofstream words_file;
    std::remove(words_filepath);
    words_file.open(words_filepath);
    words_file << "WORD_SIZE;DATA_SIZE\n";
    words_file << N << ';' << M << "\n";
    for (size_t i = 0; i < M; ++i)
        words_file << _data_vec[i].to_string() << "\n";
    words_file.close();

    if (_ham1_pairs_1.empty() || _ham1_pairs_2.empty())
        return;
    std::ofstream pairs_file;
    std::remove(pairs_filepath);
    pairs_file.open(pairs_filepath);
    pairs_file << "WORD_SIZE;PAIRS_COUNT\n";
    pairs_file << N << ';' << _ham1_pairs_1.size() << "\n";
    for (size_t i = 0; i < _ham1_pairs_1.size(); ++i)
        pairs_file << _ham1_pairs_1[i].to_string() << ';' << _ham1_pairs_2[i].to_string() << "\n";
    pairs_file.close();

    std::cout << "Saving Data successful!" << std::endl << std::endl;
}

////////////////////////////////////////////////////////////////////////////////
// finding pairs with hamming distance 1 on CPU
template<size_t N>
void find_ham1(const typename std::vector<std::bitset<N>>& _data_vec, \
    typename std::vector<std::bitset<N>>& _ham1_pairs_1, typename std::vector<std::bitset<N>>& _ham1_pairs_2, \
    const bool timeCount, const bool pairsOutput)
{
    std::chrono::steady_clock::time_point start, finish;
    std::chrono::duration<double> elapsed;

    std::cout << "Looking for pairs with hamming distance 1 ...\n";

    _ham1_pairs_1.clear();
    _ham1_pairs_2.clear();

    // Record start time
    if (timeCount) start = std::chrono::high_resolution_clock::now();

    unsigned int ham1 = 0;
    for (auto it1 = std::begin(_data_vec); it1 != std::end(_data_vec); ++it1)
    {
        for (auto it2 = std::next(it1); it2 != std::end(_data_vec); ++it2)
        {
            if (1 == hamming_distance<N>(*it1, *it2))
            {
                _ham1_pairs_1.emplace_back(std::bitset<N>(*it1));
                _ham1_pairs_2.emplace_back(std::bitset<N>(*it2));
                ++ham1;
            }
        }
    }

    // Record end time
    if (timeCount) finish = std::chrono::high_resolution_clock::now();
    if (timeCount) elapsed = finish - start;

    std::cout << "Finished!\n";
    if (timeCount) std::cout << "Elapsed time: " << elapsed.count() << " s\n";
    std::cout << ham1 << " pairs found\n\n";

    if (ham1 && pairsOutput)
    {
        std::cout << "Pairs found:\n";

        for (auto it1 = std::begin(_ham1_pairs_1), it2 = std::begin(_ham1_pairs_2); it1 != std::end(_ham1_pairs_1); ++it1, ++it2)
        {
            std::cout << *it1 << " " << *it2 << std::endl;
        }

        std::cout << std::endl;
    }
}

////////////////////////////////////////////////////////////////////////////////
// hamming distance function
template<size_t N>
unsigned int hamming_distance(const typename std::bitset<N>& A, const typename std::bitset<N>& B)
{
    return (A ^ B).count();
}

////////////////////////////////////////////////////////////////////////////////
// move data to gpu
template<size_t N, size_t M>
thrust::device_vector<unsigned int> move_data_to_GPU(const typename std::vector<std::bitset<N>>& data_vec)
{
    //N - WORD_SIZE, M - DATA_SIZE
    thrust::host_vector<unsigned int> h_words(M * SUBWORDS_PER_WORD(N));
    thrust::device_vector<unsigned int> d_words;

    std::chrono::steady_clock::time_point start, finish;
    std::chrono::duration<double> elapsed;

    // Record start time
    start = std::chrono::high_resolution_clock::now();

    int i = 0;
    for (const auto& word_bitset : data_vec)
    {
        //std::cout << std::endl << "Original " << word_bitset.to_string() << std::endl;
        if (N < UINT_BITSIZE)
        {
            std::string subword_str = word_bitset.to_string().substr(0, N);
            for (size_t subword_str_size = N; subword_str_size < UINT_BITSIZE; ++subword_str_size)
                subword_str += "0";
            unsigned int subword = (unsigned int)(std::bitset<N>(subword_str).to_ulong());
            //std::cout << "Subword: " << subword_str << ", " << subword << std::endl;
            h_words[i++] = subword;
            continue;
        }
        size_t j = 0;
        for (; j + UINT_BITSIZE < N; j += UINT_BITSIZE)
        {
            std::string subword_str = word_bitset.to_string().substr(j, UINT_BITSIZE);
            unsigned int subword = (unsigned int)(std::bitset<N>(subword_str).to_ulong());
            //std::cout << "Subword: " << subword_str << ", " << subword << std::endl;
            h_words[i++] = subword;
        }
        if (j + UINT_BITSIZE != N) // last subword smaller than UINT_BITSIZE
        {
            std::string subword_str = word_bitset.to_string().substr(j, N - j);
            for (size_t subword_str_size = N - j; subword_str_size < UINT_BITSIZE; ++subword_str_size)
                subword_str += "0";
            unsigned int subword = (unsigned int)(std::bitset<N>(subword_str).to_ulong());
            //std::cout << "Subword: " << subword_str << ", " << subword << std::endl;
            h_words[i++] = subword;
        }
    }
    d_words = h_words;

    finish = std::chrono::high_resolution_clock::now();
    elapsed = finish - start;

    std::cout << std::endl << "Data moved to GPU" << std::endl;
    std::cout << "Elapsed time: " << elapsed.count() << " s" << std::endl << std::endl;
    return d_words;
}

////////////////////////////////////////////////////////////////////////////////
// HammingOne kernel
__global__ void find_ham1_GPU_ker(const unsigned int* subwords, unsigned int* pair_flags, const unsigned int subwords_per_pair_flags)
{
    const unsigned int word_idx = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int subwords_per_word = SUBWORDS_PER_WORD(WORD_SIZE);

    if (word_idx >= DATA_SIZE)
        return;

    unsigned int* word = new unsigned int[subwords_per_word];
    for (size_t i = 0; i < subwords_per_word; ++i)
    {
        word[i] = subwords[word_idx * subwords_per_word + i];
    }

    unsigned int hamming_distance, flag_subword_offset, flag_in_subword;

    for (size_t comparison_idx = word_idx + 1; comparison_idx < DATA_SIZE; ++comparison_idx)
    {
        hamming_distance = 0;
        for (size_t i = 0; i < subwords_per_word && hamming_distance < 2; ++i)
        {
            hamming_distance += __popc(word[i] ^ subwords[comparison_idx * subwords_per_word + i]);
        }
        if (!(hamming_distance >> 1)) // true when hamming_distance == 1
        {
            flag_subword_offset = comparison_idx / UINT_BITSIZE;
            flag_in_subword = 1 << UINT_BITSIZE - 1 - comparison_idx % UINT_BITSIZE;
            pair_flags[word_idx * subwords_per_pair_flags + flag_subword_offset] |= flag_in_subword;
        }
    }

    delete[] word;
}

////////////////////////////////////////////////////////////////////////////////
// Counting kernel - very similar time on CPU
//__global__ void count_ones(unsigned int* d_data, size_t pair_flags_size)
//{
//    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
//    if (tid >= pair_flags_size)
//        return;
//    d_data[tid] = __popc(d_data[tid]);
//}

////////////////////////////////////////////////////////////////////////////////
// finding pairs with hamming distance 1 on GPU
template<size_t N>
void find_ham1_GPU(thrust::device_vector<unsigned int>& d_subwords, \
    thrust::device_vector<unsigned int>& d_pair_flags, \
    thrust::host_vector<unsigned int>& h_pair_flags, size_t pair_flags_size, \
    const bool timeCount, const bool pairsOutput, const typename std::vector<std::bitset<N>>& _data_vec)
{
    unsigned int threads = 512;
    unsigned int blocks = (unsigned int)std::ceil(DATA_SIZE / (double)threads);
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    unsigned int pairs_count = 0, pairs_count_GPU = 0;
    const unsigned int subwords_per_pair_flags = pair_flags_size / DATA_SIZE;
    auto d_subwords_ptr = thrust::raw_pointer_cast(d_subwords.begin().base());
    auto d_pair_flags_ptr = thrust::raw_pointer_cast(d_pair_flags.begin().base());
    float elapsed;
    hipEvent_t start, stop;

    if (timeCount) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    std::cout << "Looking for pairs with hamming distance 1 ...\n";

    if (timeCount) hipEventRecord(start, 0);
    find_ham1_GPU_ker<<<dimGrid, dimBlock>>>(d_subwords_ptr, d_pair_flags_ptr, subwords_per_pair_flags);
    if (timeCount) hipEventRecord(stop, 0);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));

    hipDeviceSynchronize();

    if (timeCount) hipEventElapsedTime(&elapsed, start, stop);

    std::cout << "Finished!\n";
    if (timeCount) std::cout << "Elapsed time: " << elapsed << " ms\n";

    h_pair_flags = d_pair_flags;
    for (size_t i = 0; i < pair_flags_size; ++i)
    {
        pairs_count += __popcnt(h_pair_flags[i]);
    }
    std::cout << pairs_count << " pairs found\n\n";

    // Te time is basically the same on CPU
    //int threads2 = 512;
    //int blocks2 = (unsigned int)std::ceil(pair_flags_size / (double)threads);
    //dim3 dimBlock2(threads2, 1, 1);
    //dim3 dimGrid2(blocks2, 1, 1);
    //count_ones<<<dimGrid2, dimBlock2>>>(thrust::raw_pointer_cast(d_pair_flags.begin().base()), pair_flags_size);
    //hipDeviceSynchronize();
    //h_pair_flags = d_pair_flags;
    //for (size_t i = 0; i < pair_flags_size; ++i)
    //{
    //    pairs_count_GPU += h_pair_flags[i];
    //}
    //
    //std::cout << pairs_count_GPU << " pairs found on GPU\n\n";

    if (pairs_count && pairsOutput)
        print_pairs_from_flags<N>(h_pair_flags, pair_flags_size, _data_vec);

    if (timeCount) {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    thrust::fill(d_pair_flags.begin(), d_pair_flags.end(), 0);
}

////////////////////////////////////////////////////////////////////////////////
// pairs_flag to pairs output
template<size_t N>
void print_pairs_from_flags(thrust::host_vector<unsigned int>& h_pair_flags, size_t pair_flags_size, \
    const typename std::vector<std::bitset<N>>& data_vec)
{
    const unsigned int subwords_per_word_flags = pair_flags_size / DATA_SIZE;

    std::cout << "Pairs found:\n";

    for (size_t word_idx = 0; word_idx < DATA_SIZE; ++word_idx)
    {
        bool flag_found = false;
        unsigned int* word_flags = new unsigned int[subwords_per_word_flags];
        for (size_t i = 0; i < subwords_per_word_flags; ++i)
        {
            word_flags[i] = h_pair_flags[word_idx * subwords_per_word_flags + i];
        }
        for (size_t i = 0; i < subwords_per_word_flags; ++i)
            if (word_flags[i]) {
                flag_found = true;
                break;
            }
        if (!flag_found) continue;
        for (int i = subwords_per_word_flags-1; i >= 0; --i)
        {
            if (!word_flags[i])
                continue;
            int flags_set = __popcnt(word_flags[i]);
            int flag_pos = (i+1) * UINT_BITSIZE - 1;
            size_t j = 0;
            while (j < flags_set)
            {
                if (word_flags[i] % 2) {
                    std::cout << data_vec[word_idx] << " " << data_vec[flag_pos] << std::endl;
                    ++j;
                }
                word_flags[i] = word_flags[i] >> 1;
                --flag_pos;
            }
        }
        delete[] word_flags;
    }

    std::cout << std::endl;
}

////////////////////////////////////////////////////////////////////////////////
int main()
{
    bool updated_data_GPU = true;
    unsigned short menu_choice = 0;
    size_t pair_flags_size = DATA_SIZE * (std::ceil((double)DATA_SIZE / (double)UINT_BITSIZE));
    thrust::device_vector<unsigned int> d_subwords;
    thrust::device_vector<unsigned int> d_pair_flags(pair_flags_size, 0);
    thrust::host_vector<unsigned int> h_pair_flags;
    std::vector<std::bitset<WORD_SIZE>> data_vec;
    std::vector<std::bitset<WORD_SIZE>> ham1_pairs_1;
    std::vector<std::bitset<WORD_SIZE>> ham1_pairs_2;

    while (menu_choice != 5) {
        std::cout << "1. Generate Data" << std::endl;
        std::cout << "2. Save/Load Data" << std::endl;
        if (!data_vec.empty()) {
            if (d_subwords.empty())
                std::cout << "3. Move Data to GPU - !!! No Data on GPU !!!" << std::endl;
            else if (!updated_data_GPU)
                std::cout << "3. Move Data to GPU - !!! Data on GPU not matching Data on CPU !!!" << std::endl;
            else
                std::cout << "3. Move Data to GPU" << std::endl;
            std::cout << "4. Find Pairs" << std::endl;
        }
        else {
            std::cout << "3. Move Data to GPU - !!! Generate/Load Data before attempting to move the data to GPU !!!" << std::endl;
            std::cout << "4. Find Pairs - !!! Generate/Load Data before attempting to find pairs !!!" << std::endl;
        }
        std::cout << "5. Exit" << std::endl;
        std::cout << "6. Clear Console" << std::endl;
        std::cout << "Choice: ";
        std::cin >> menu_choice;
        switch (menu_choice)
        {
        case 1:
            if (!data_vec.empty())
                data_vec.clear();
            data_vec.reserve(DATA_SIZE);
            if (!ham1_pairs_1.empty()) {
                ham1_pairs_1.clear();
                ham1_pairs_2.clear();
            }
            std::cout << std::endl;
            generate_data<WORD_SIZE, DATA_SIZE>(data_vec);
            updated_data_GPU = false;
            break;
        case 2:
            std::cout << std::endl;
            while (menu_choice != 3)
            {
                std::cout << "1. Save Data" << std::endl;
                std::cout << "2. Load Data" << std::endl;
                std::cout << "3. Go Back" << std::endl;
                std::cout << "Choice: ";
                std::cin >> menu_choice;
                std::cout << std::endl;
                switch (menu_choice)
                {
                case 1:
                    save_data<WORD_SIZE, DATA_SIZE>("./words_data.csv", "./pairs_data.csv", data_vec, ham1_pairs_1, ham1_pairs_2);
                    break;
                case 2:
                    load_data<WORD_SIZE, DATA_SIZE>("./words_data.csv", "./pairs_data.csv", data_vec, ham1_pairs_1, ham1_pairs_2);
                    updated_data_GPU = false;
                    break;
                case 3:
                    break;
                default:
                    std::cout << "Please provide a valid choice" << std::endl << std::endl;
                    break;
                }
            }
            menu_choice = 2;
            break;
        case 3:
            if (!data_vec.empty()) {
                d_subwords = move_data_to_GPU<WORD_SIZE, DATA_SIZE>(data_vec);
                updated_data_GPU = true;
            }
            else
                std::cout << std::endl << "!!! Generate / Load Data before attempting to move the data to GPU !!!" << std::endl << std::endl;
            break;
        case 4:
            std::cout << std::endl;
            if (!data_vec.empty()) {
                while (menu_choice != 3)
                {
                    std::cout << "1. Use CPU" << std::endl;
                    if (d_subwords.empty())
                        std::cout << "2. Use GPU - !!! No Data on GPU !!!" << std::endl;
                    else if (!updated_data_GPU)
                        std::cout << "2. Use GPU - !!! Data on GPU not matching Data on CPU !!!" << std::endl;
                    else
                        std::cout << "2. Use GPU" << std::endl;
                    std::cout << "3. Go Back" << std::endl;
                    std::cout << "Choice: ";
                    std::cin >> menu_choice;
                    std::cout << std::endl;
                    switch (menu_choice)
                    {
                    case 1:
                        char c;
                        do {
                            std::cout << "Output pairs to console? (y/n):";
                            std::cin.clear();
                            std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
                            c = std::getc(stdin);
                            if (c == 'y' || c == 'Y') {
                                find_ham1<WORD_SIZE>(data_vec, ham1_pairs_1, ham1_pairs_2, true, true);
                                break;
                            }
                            else if (c == 'n' || c == 'N') {
                                find_ham1<WORD_SIZE>(data_vec, ham1_pairs_1, ham1_pairs_2, true, false);
                                break;
                            }
                            std::cout << "Please provide a valid choice" << std::endl;
                        } while (true);
                        break;
                    case 2:
                        if (d_subwords.empty())
                            std::cout << std::endl << "!!! No Data on GPU !!!" << std::endl << std::endl;
                        else do {
                            std::cout << "Output pairs to console? (y/n):";
                            std::cin.clear();
                            std::cin.ignore(std::numeric_limits<std::streamsize>::max(), '\n');
                            c = std::getc(stdin);
                            if (c == 'y' || c == 'Y') {
                                find_ham1_GPU<WORD_SIZE>(d_subwords, d_pair_flags, h_pair_flags, pair_flags_size, true, true, data_vec);
                                break;
                            }
                            else if (c == 'n' || c == 'N') {
                                find_ham1_GPU<WORD_SIZE>(d_subwords, d_pair_flags, h_pair_flags, pair_flags_size, true, false, data_vec);
                                break;
                            }
                            std::cout << "Please provide a valid choice" << std::endl;
                        } while (true);
                        break;
                    case 3:
                        break;
                    default:
                        std::cout << "Please provide a valid choice" << std::endl << std::endl;
                        break;
                    }
                }
            }
            else
                std::cout << std::endl << "!!! Generate/Load Data before attempting to find pairs !!!" << std::endl << std::endl;
            menu_choice = 4;
            break;
        case 5:
            break;
        case 6:
            system("CLS");
            break;
        default:
            std::cout << std::endl << "Please provide a valid choice" << std::endl << std::endl;
            break;
        }
    }

    return 0;
}